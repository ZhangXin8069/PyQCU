#include "../python/pyqcu.h"
#include "../include/qcu.h"
#pragma optimize(5)
using namespace qcu;
void applyWilsonBistabCgDslashQcu(long long _fermion_out, long long _fermion_in, long long _gauge, long long _set_ptrs, long long _params)
{
    hipDeviceSynchronize();
    void *fermion_out = (void *)_fermion_out;
    void *fermion_in = (void *)_fermion_in;
    void *gauge = (void *)_gauge;
    void *set_ptrs = (void *)_set_ptrs;
    void *params = (void *)_params;
    int set_index = static_cast<int *>(params)[_SET_INDEX_];
    int data_type = static_cast<int *>(params)[_DATA_TYPE_];
    if (data_type == _LAT_C64_)
    {
        // define for apply_wilson_dslash
        LatticeSet<float> *set_ptr = static_cast<LatticeSet<float> *>((void *)(static_cast<long long *>(set_ptrs)[set_index]));
        // dptzyxcc2ccdptzyx<float>(gauge, &_set);
        // tzyxsc2sctzyx<float>(fermion_in, &_set);
        // tzyxsc2sctzyx<float>(fermion_out, &_set);
        LatticeWilsonDslash<float> _wilson_dslash;
        _wilson_dslash.give(set_ptr);
        // { // test
        //     printf("fermion_out: %p\n", fermion_out);
        //     printf("fermion_in: %p\n", fermion_in);
        //     printf("gauge: %p\n", gauge);
        // }
        {
            void *device_vec0, *device_vec1, *device_vals;
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
            checkCudaErrors(hipMallocAsync(
                &device_vec0, set_ptr->lat_4dim_SC * sizeof(LatticeComplex<float>),
                set_ptr->stream));
            checkCudaErrors(hipMallocAsync(
                &device_vec1, set_ptr->lat_4dim_SC * sizeof(LatticeComplex<float>),
                set_ptr->stream));
            checkCudaErrors(hipMallocAsync(
                &device_vals, _vals_size_ * sizeof(LatticeComplex<float>), set_ptr->stream));
            give_1custom<float><<<1, 1, 0, set_ptr->stream>>>(
                device_vals, _lat_4dim_, float(set_ptr->lat_4dim), 0.0);
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
            // src_o-set_ptr->kappa()**2*dslash_oe(dslash_eo(src_o))
            _wilson_dslash.run_eo(device_vec0, fermion_in, gauge);
            _wilson_dslash.run_oe(device_vec1, device_vec0, gauge);
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
            bistabcg_give_dest_o<float><<<set_ptr->gridDim, set_ptr->blockDim, 0,
                                          set_ptr->stream>>>(
                fermion_out, fermion_in, device_vec1, set_ptr->kappa(), device_vals);
            checkCudaErrors(hipFreeAsync(device_vec0, set_ptr->stream));
            checkCudaErrors(hipFreeAsync(device_vec1, set_ptr->stream));
            checkCudaErrors(hipFreeAsync(device_vals, set_ptr->stream));
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        }
        // ccdptzyx2dptzyxcc<float>(gauge, &_set);
        // sctzyx2tzyxsc<float>(fermion_in, &_set);
        // sctzyx2tzyxsc<float>(fermion_out, &_set);
    }
    else if (data_type == _LAT_C128_)
    {
        // define for apply_wilson_dslash
        LatticeSet<double> *set_ptr = static_cast<LatticeSet<double> *>((void *)(static_cast<long long *>(set_ptrs)[set_index]));
        // dptzyxcc2ccdptzyx<double>(gauge, &_set);
        // tzyxsc2sctzyx<double>(fermion_in, &_set);
        // tzyxsc2sctzyx<double>(fermion_out, &_set);
        LatticeWilsonDslash<double> _wilson_dslash;
        _wilson_dslash.give(set_ptr);
        // { // test
        //     printf("fermion_out: %p\n", fermion_out);
        //     printf("fermion_in: %p\n", fermion_in);
        //     printf("gauge: %p\n", gauge);
        // }
        {
            void *device_vec0, *device_vec1, *device_vals;
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
            checkCudaErrors(hipMallocAsync(
                &device_vec0, set_ptr->lat_4dim_SC * sizeof(LatticeComplex<double>),
                set_ptr->stream));
            checkCudaErrors(hipMallocAsync(
                &device_vec1, set_ptr->lat_4dim_SC * sizeof(LatticeComplex<double>),
                set_ptr->stream));
            checkCudaErrors(hipMallocAsync(
                &device_vals, _vals_size_ * sizeof(LatticeComplex<double>), set_ptr->stream));
            give_1custom<double><<<1, 1, 0, set_ptr->stream>>>(
                device_vals, _lat_4dim_, double(set_ptr->lat_4dim), 0.0);
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
            // src_o-set_ptr->kappa()**2*dslash_oe(dslash_eo(src_o))
            _wilson_dslash.run_eo(device_vec0, fermion_in, gauge);
            _wilson_dslash.run_oe(device_vec1, device_vec0, gauge);
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
            bistabcg_give_dest_o<double><<<set_ptr->gridDim, set_ptr->blockDim, 0,
                                           set_ptr->stream>>>(
                fermion_out, fermion_in, device_vec1, set_ptr->kappa(), device_vals);
            checkCudaErrors(hipFreeAsync(device_vec0, set_ptr->stream));
            checkCudaErrors(hipFreeAsync(device_vec1, set_ptr->stream));
            checkCudaErrors(hipFreeAsync(device_vals, set_ptr->stream));
            checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        }
        // ccdptzyx2dptzyxcc<double>(gauge, &_set);
        // sctzyx2tzyxsc<double>(fermion_in, &_set);
        // sctzyx2tzyxsc<double>(fermion_out, &_set);
    }
    else
    {
        printf("data_type error\n");
    }
    hipDeviceSynchronize();
}