#include "../python/pyqcu.h"
#include "../include/qcu.h"
#pragma optimize(5)
using namespace qcu;
using T = float;
void testWilsonDslashQcu(long long _fermion_out, long long _fermion_in, long long _gauge, long long _set_ptrs, long long _params)
{
    hipDeviceSynchronize();
    void *fermion_out = (void *)_fermion_out;
    void *fermion_in = (void *)_fermion_in;
    void *gauge = (void *)_gauge;
    void *set_ptrs = (void *)_set_ptrs;
    void *params = (void *)_params;
    int set_index = static_cast<int *>(params)[_SET_INDEX_];
    // define for test_wilson_dslash
    LatticeSet<T> *set_ptr = static_cast<LatticeSet<T> *>((void *)(static_cast<long long *>(set_ptrs)[set_index]));
    // dptzyxcc2ccdptzyx<T>(gauge, &_set);
    // tzyxsc2sctzyx<T>(fermion_in, &_set);
    // tzyxsc2sctzyx<T>(fermion_out, &_set);
    auto start = std::chrono::high_resolution_clock::now();
    wilson_dslash<T><<<set_ptr->gridDim, set_ptr->blockDim>>>(gauge, fermion_in, fermion_out,
                                                              set_ptr->device_params);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    hipError_t err = hipGetLastError();
    checkCudaErrors(err);
    printf("wilson dslash total time: (without malloc free memcpy) :%.9lf "
           "sec\n",
           double(duration) / 1e9);
    // ccdptzyx2dptzyxcc<T>(gauge, &_set);
    // sctzyx2tzyxsc<T>(fermion_in, &_set);
    // sctzyx2tzyxsc<T>(fermion_out, &_set);
    hipDeviceSynchronize();
}
void testCloverDslashQcu(long long _fermion_out, long long _fermion_in, long long _gauge, long long _set_ptrs, long long _params)
{
    hipDeviceSynchronize();
    void *fermion_out = (void *)_fermion_out;
    void *fermion_in = (void *)_fermion_in;
    void *gauge = (void *)_gauge;
    void *set_ptrs = (void *)_set_ptrs;
    void *params = (void *)_params;
    int set_index = static_cast<int *>(params)[_SET_INDEX_];
    // define for test_clover_dslash
    LatticeSet<T> *set_ptr = static_cast<LatticeSet<T> *>((void *)(static_cast<long long *>(set_ptrs)[set_index]));
    // dptzyxcc2ccdptzyx<T>(gauge, &_set);
    // tzyxsc2sctzyx<T>(fermion_in, &_set);
    // tzyxsc2sctzyx<T>(fermion_out, &_set);
    LatticeWilsonDslash<T> _wilson_dslash;
    _wilson_dslash.give(set_ptr);
    void *clover;
    checkCudaErrors(hipMallocAsync(
        &clover, (set_ptr->lat_4dim * _LAT_SCSC_) * sizeof(LatticeComplex<T>),
        set_ptr->stream));
    hipError_t err;
    {
        // wilson dslash
        _wilson_dslash.run_test(fermion_out, fermion_in, gauge);
    }
    {
        // make clover
        checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        auto start = std::chrono::high_resolution_clock::now();
        make_clover<T><<<set_ptr->gridDim, set_ptr->blockDim, 0, set_ptr->stream>>>(
            gauge, clover, set_ptr->device_params);
        checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        auto end = std::chrono::high_resolution_clock::now();
        auto duration =
            std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
                .count();
        err = hipGetLastError();
        checkCudaErrors(err);
        printf("make clover total time: (without malloc free memcpy) :%.9lf sec\n ",
               double(duration) / 1e9);
    }
    {
        // inverse clover
        checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        auto start = std::chrono::high_resolution_clock::now();
        inverse_clover<T><<<set_ptr->gridDim, set_ptr->blockDim, 0, set_ptr->stream>>>(
            clover, set_ptr->device_params);
        checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        auto end = std::chrono::high_resolution_clock::now();
        auto duration =
            std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
                .count();
        err = hipGetLastError();
        checkCudaErrors(err);
        printf(
            "inverse clover total time: (without malloc free memcpy) :%.9lf sec\n ",
            double(duration) / 1e9);
    }
    {
        // give clover
        checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        auto start = std::chrono::high_resolution_clock::now();
        give_clover<T><<<set_ptr->gridDim, set_ptr->blockDim, 0, set_ptr->stream>>>(
            clover, fermion_out, set_ptr->device_params);
        checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
        auto end = std::chrono::high_resolution_clock::now();
        auto duration =
            std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
                .count();
        err = hipGetLastError();
        checkCudaErrors(err);
        printf("give clover total time: (without malloc free memcpy) :%.9lf sec\n ",
               double(duration) / 1e9);
    }
    // ccdptzyx2dptzyxcc<T>(gauge, &_set);
    // sctzyx2tzyxsc<T>(fermion_in, &_set);
    // sctzyx2tzyxsc<T>(fermion_out, &_set);
    // free
    checkCudaErrors(hipFreeAsync(clover, set_ptr->stream));
    checkCudaErrors(hipStreamSynchronize(set_ptr->stream));
    hipDeviceSynchronize();
}
