
#include <hip/hip_runtime.h>
// #include "../python/pyqcu.h"
// #include "../include/qcu.h"
// #pragma optimize(5)
// using namespace qcu;
// void applyWilsonBistabCgQcu(long long _fermion_out, long long _fermion_in, long long _gauge, long long _set_ptrs, long long _params)
// {
//   cudaDeviceSynchronize();
//   void *fermion_out = (void *)_fermion_out;
//   void *fermion_in = (void *)_fermion_in;
//   void *gauge = (void *)_gauge;
//   void *set_ptrs = (void *)_set_ptrs;
//   void *params = (void *)_params;
//   int set_index = static_cast<int *>(params)[_SET_INDEX_];
//   int data_type = static_cast<int *>(params)[_DATA_TYPE_];
//   if (data_type == _LAT_C64_)
//   {
//     LatticeSet<float> *set_ptr = static_cast<LatticeSet<float> *>((void *)(static_cast<long long *>(set_ptrs)[set_index])); // define for apply_wilson_bistabcg
//     // dptzyxcc2ccdptzyx<float>(gauge, &_set);
//     // ptzyxsc2psctzyx<float>(fermion_in, &_set);
//     // ptzyxsc2psctzyx<float>(fermion_out, &_set);
//     LatticeWilsonBistabCg<float> _bistabcg;
//     _bistabcg.give(set_ptr);
//     _bistabcg.init(fermion_out, fermion_in, gauge);
//     if (set_ptr->host_params[_VERBOSE_])
//     {
//       _bistabcg.run_test();
//     }
//     else
//     {
//       _bistabcg.run();
//     }
//     _bistabcg.end();
//     // ccdptzyx2dptzyxcc<float>(gauge, &_set);
//     // psctzyx2ptzyxsc<float>(fermion_in, &_set);
//     // psctzyx2ptzyxsc<float>(fermion_out, &_set);
//   }
//   else if (data_type == _LAT_C128_)
//   {
//     LatticeSet<double> *set_ptr = static_cast<LatticeSet<double> *>((void *)(static_cast<long long *>(set_ptrs)[set_index])); // define for apply_wilson_bistabcg
//     // dptzyxcc2ccdptzyx<double>(gauge, &_set);
//     // ptzyxsc2psctzyx<double>(fermion_in, &_set);
//     // ptzyxsc2psctzyx<double>(fermion_out, &_set);
//     LatticeWilsonBistabCg<double> _bistabcg;
//     _bistabcg.give(set_ptr);
//     _bistabcg.init(fermion_out, fermion_in, gauge);
//     if (set_ptr->host_params[_VERBOSE_])
//     {
//       _bistabcg.run_test();
//     }
//     else
//     {
//       _bistabcg.run();
//     }
//     _bistabcg.end();
//     // ccdptzyx2dptzyxcc<double>(gauge, &_set);
//     // psctzyx2ptzyxsc<double>(fermion_in, &_set);
//     // psctzyx2ptzyxsc<double>(fermion_out, &_set);
//   }
//   else
//   {
//     printf("data_type error\n");
//   }
//   cudaDeviceSynchronize();
// }