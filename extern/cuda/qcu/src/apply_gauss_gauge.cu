#include "../python/pyqcu.h"
#include "../include/qcu.h"
#pragma optimize(5)
using namespace qcu;
void applyGaussGaugeQcu(long long _gauge, long long _set_ptrs, long long _params)
{
  hipDeviceSynchronize();
  void *gauge = (void *)_gauge;
  void *set_ptrs = (void *)_set_ptrs;
  void *params = (void *)_params;
  int set_index = static_cast<int *>(params)[_SET_INDEX_];
  int data_type = static_cast<int *>(params)[_DATA_TYPE_];
  if (data_type == _LAT_C64_)
  {
    void *set_ptr = (void *)(static_cast<long long *>(set_ptrs)[set_index]); // define for apply_gauss_gauge
    // dptzyxcc2ccdptzyx<float>(gauge, &_set);
    make_gauss_gauge<float>(gauge, set_ptr);
    // ccdptzyx2dptzyxcc<float>(gauge, &_set);
  }
  else if (data_type == _LAT_C128_)
  {
    void *set_ptr = (void *)(static_cast<long long *>(set_ptrs)[set_index]); // define for apply_gauss_gauge
    // dptzyxcc2ccdptzyx<double>(gauge, &_set);
    make_gauss_gauge<double>(gauge, set_ptr);
    // ccdptzyx2dptzyxcc<double>(gauge, &_set);
  }
  else
  {
    printf("data_type error\n");
  }
  hipDeviceSynchronize();
}