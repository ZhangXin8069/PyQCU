#include "../python/pyqcu.h"
#include "../include/qcu.h"
#pragma optimize(5)
using namespace qcu;
void applyDslashQcu(long long _fermion_out, long long _fermion_in, long long _gauge, long long _clover, long long _set_ptrs, long long _params)
{
  hipDeviceSynchronize();
  void *fermion_out = (void *)_fermion_out;
  void *fermion_in = (void *)_fermion_in;
  void *gauge = (void *)_gauge;
  void *clover = (void *)_clover;
  void *set_ptrs = (void *)_set_ptrs;
  void *params = (void *)_params;
  int set_index = static_cast<int *>(params)[_SET_INDEX_];
  int data_type = static_cast<int *>(params)[_DATA_TYPE_];
  if (data_type == _LAT_C64_)
  {
    LatticeSet<float> *set_ptr = static_cast<LatticeSet<float> *>((void *)(static_cast<long long *>(set_ptrs)[set_index])); // define for apply_clover_dslash
    // dptzyxcc2ccdptzyx<float>(gauge, &_set);
    LatticeWilsonDslash<float> _wilson_dslash;
    _wilson_dslash.give(set_ptr);
    {
      // wilson dslash
      if (set_ptr->host_params[_VERBOSE_])
      {
        _wilson_dslash.run_test(fermion_out, fermion_in, gauge);
      }
      else
      {
        _wilson_dslash.run(fermion_out, fermion_in, gauge);
      }
    }
    {
      // clover dslash
      if (set_ptr->host_params[_VERBOSE_])
      {
        printf("clover:%p\n", clover);
        printf("long long clover:%lld\n", (long long)clover);
      }
      if ((long long)clover)
      {
        LatticeCloverDslash<float> _clover_dslash;
        _clover_dslash.give(set_ptr);
        _clover_dslash.init(clover);
        // give clover
        _clover_dslash.give(fermion_out);
        _clover_dslash.end();
      }
    }
    // ccdptzyx2dptzyxcc<float>(gauge, &_set);
  }
  else if (data_type == _LAT_C128_)
  {
    LatticeSet<double> *set_ptr = static_cast<LatticeSet<double> *>((void *)(static_cast<long long *>(set_ptrs)[set_index])); // define for apply_clover_dslash
    // dptzyxcc2ccdptzyx<double>(gauge, &_set);
    LatticeWilsonDslash<double> _wilson_dslash;
    _wilson_dslash.give(set_ptr);
    {
      // wilson dslash
      if (set_ptr->host_params[_VERBOSE_])
      {
        _wilson_dslash.run_test(fermion_out, fermion_in, gauge);
      }
      else
      {
        _wilson_dslash.run(fermion_out, fermion_in, gauge);
      }
    }
    {
      // clover dslash
      if (set_ptr->host_params[_VERBOSE_])
      {
        printf("clover:%p\n", clover);
        printf("long long clover:%lld\n", (long long)clover);
      }
      if ((long long)clover)
      {
        LatticeCloverDslash<double> _clover_dslash;
        _clover_dslash.give(set_ptr);
        _clover_dslash.init(clover);
        // give clover
        _clover_dslash.give(fermion_out);
        _clover_dslash.end();
      }
    }
    // ccdptzyx2dptzyxcc<double>(gauge, &_set);
  }
  else
  {
    printf("data_type error\n");
  }
  hipDeviceSynchronize();
}